#include "hip/hip_runtime.h"
//
// Created by Xiaobo Wu on 2021/2/26.
//

#include "kmeans.h"
#include "kmeans.cpp"
#include "dataloader.h"
#include "dataloader.cpp"
#include "mr_coreset.cu"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>
#include <time.h>


using namespace std;
using namespace coreset;


int main(){
    clock_t start,end;
    DataLoader<float> dataloader(2); //dimension = 6
    vector<float> data = dataloader.Loader_1D("denmark-latest.csv");
    //vector<vector<float> > sampled_data = dataloader.DataSample(data, 1000);  // sample 1000 items of data
    
    /*KMeans kmeans(8, "k-means++",10, 10);
    vector<vector<float> > init_centers = kmeans.KMeans_pp_Init(data, 5);
    for(int i=0; i<init_centers.size(); i++){
        vector<float> center = init_centers[i];
	for(int j=0; j<center.size(); j++)
	    cout<<center[j]<<" ";
	cout<<endl;
    }*/

    //Test Kmeans method:
    //kmeans.Fit(sampled_data);
    // cout<<kmeans.GetCenters().size()<<endl;
    // cout<<kmeans.GetCost()<<endl;
    // vector<int> labels=kmeans.GetLabel();
    // for(int i=0; i<labels.size(); i++)
    // {
    //     cout<<labels[i]<<endl;
    // }


    //Test Coreset method:
    //thrust::device_vector<float> device_points(data.begin(), data.end());
    unsigned int dimension = dataloader.dimension;
    unsigned int n_cluster = 5;
    size_int n = data.size() / dimension; 
    //float centers[n_cluster * dimension];
    vector<float> data_weights(n, 1.0);
    //k_means_pp_init_cu(points, n, centers, n_cluster, dimension);
   
    unsigned int n_coreset = 20000;
    //coreset::FlatPoints coreset(n_coreset, dimension);
    coreset::Points coreset(n_coreset, dimension);
    start = clock();
    coreset = compute_coreset(data, data_weights, dimension, n_cluster, n_coreset);
    end = clock();
    cout<<"time = "<<double(end-start)/CLOCKS_PER_SEC<<"s"<<endl;
    //coreset = compute_coreset_mr(data, data_weights, dimension, n_cluster, n_coreset, 30);
   /*
    vector<float> v = coreset.GetValues();
    vector<float> w = coreset.GetWeights();
    
    size_int ind = 0;
    for(int i=0; i<n_coreset; i++){
        for(int j=0; j<dimension; j++){
	    cout<<v[ind]<<" ";
	    ind ++;
	}
	cout<<endl;
    }
    
    for(int i=0; i<n_coreset; i++){
        cout<<w[i]<<endl;
    }
    */
    /*
    vector<vector<float> > v = coreset.GetValues();
    vector<float> w = coreset.GetWeights();
    for(int i=0; i<n_coreset; i++){
	for(int j=0; j<dimension; j++){
		cout<<v[i][j]<<" ";
	}		
        cout<<w[i]<<endl;
    }*/
    

    return 0;
}


